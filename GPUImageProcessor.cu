#include "hip/hip_runtime.h"
#include "GPUImageProcessor.h"
#include "hip/hip_runtime.h"
#include ""

// Kernel Functions...
__global__ void colorToGray(const uchar* input, uchar* d_gray, const int rowLength, const int colLength, const int colorFlag) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * rowLength + col;

    float grayval = 0;
    if (row < colLength && col < rowLength) {
        if (colorFlag == 1) {
            d_gray[idx] += static_cast<uchar>(0.21 * input[idx]);
        }
        else if (colorFlag == 2) {
            d_gray[idx] += static_cast<uchar>(0.07 * input[idx]);
        }
        else {
            d_gray[idx] += static_cast<uchar>(0.72 * input[idx]);

        }
        //d_gray[idx] = static_cast<uchar>(0.21 * red[idx] + 0.07 * blue[idx] + 0.72 * green[idx]);
    }
}

//__constant__ const int halfKernel = 1;

__global__ void convolution(const uchar* input, uchar* output, const uchar* g_kernel, const int kernelDim,
                             const int rows, const int cols, const float reduction, const int sobelFlag = 0, uchar* mask = 0) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;
    const int halfKernel = kernelDim / 2;

    __shared__ uchar kernel[9];

    if (threadIdx.x < 9) {
        kernel[threadIdx.x] = g_kernel[threadIdx.x];
    } 
    __syncthreads();
    
    if (row < rows && col < cols) {
        if (row < halfKernel || col < halfKernel || row + halfKernel > rows || col + halfKernel > cols) {
            // edge case, do nothing for now
            output[idx] = input[idx];
        }
        else {
            float sum = 0;
            // i iterates over the columns, j iterates over the rows
            if (kernelDim == 9 && !sobelFlag) {
                sum += input[idx - 1 * cols - 1] * kernel[(- 1 + halfKernel) * kernelDim + (- 1 + halfKernel)]; // -1 -1
                sum += input[idx  * cols - 1] * kernel[(halfKernel) * kernelDim + (-1 + halfKernel)];           //  0 -1
                sum += input[idx + 1 * cols - 1] * kernel[(1 + halfKernel) * kernelDim + (-1 + halfKernel)];    //  1 -1
                sum += input[idx -1 * cols] * kernel[(-1 + halfKernel) * kernelDim + (halfKernel)];             // -1  0
                sum += input[idx * cols] * kernel[(halfKernel) * kernelDim + (halfKernel)];     //  0  0
                sum += input[idx + 1 * cols] * kernel[(1 + halfKernel) * kernelDim + (halfKernel)];             //  1  0
                sum += input[idx - 1 * cols + 1] * kernel[(-1 + halfKernel) * kernelDim + (1 + halfKernel)];    // -1  1
                sum += input[idx* cols + 1] * kernel[(halfKernel) * kernelDim + (1 + halfKernel)];              //  0  1
                sum += input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];     //  1  1
            }
            else if (sobelFlag) {
                if (sobelFlag == 1) {
                    sum -= input[idx - 1 * cols - 1] * kernel[(-1 + halfKernel) * kernelDim + (-1 + halfKernel)]; 
                } else {
                    sum += input[idx - 1 * cols - 1] * kernel[(-1 + halfKernel) * kernelDim + (-1 + halfKernel)]; 
                }
                if (sobelFlag == 2) {
                    sum += input[idx + 0 * cols - 1] * kernel[(halfKernel)*kernelDim + (-1 + halfKernel)];          
                }
                sum += input[idx + 1 * cols - 1] * kernel[(1 + halfKernel) * kernelDim + (-1 + halfKernel)];   
                if (sobelFlag == 1) {
                    sum -= input[idx - 1 * cols] * kernel[(-1 + halfKernel) * kernelDim + (halfKernel)]; 
                }
                if (sobelFlag == 1) {
                    sum += input[idx + 1 * cols] * kernel[(1 + halfKernel) * kernelDim + (halfKernel)];
                }
                sum -= input[idx - 1 * cols + 1] * kernel[(-1 + halfKernel) * kernelDim + (1 + halfKernel)];   
                if (sobelFlag == 2) {
                    sum -= input[idx + 0 * cols + 1] * kernel[(halfKernel)*kernelDim + (1 + halfKernel)];          
                }
                if (sobelFlag == 2) {
                    sum -= input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];   
                } else {
                    sum += input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];   
                }
            }
            // TODO potentially remove this section
            else {
                for (int i = -halfKernel; i <= halfKernel; i++) {
                    for (int j = -halfKernel; j <= halfKernel; j++) {
                        // Gx kernel when sobel == 1, Gy kernel when sobel == 2
                        if ((sobelFlag == 1 && j == -halfKernel) || (sobelFlag == 2 && i == halfKernel)) {
                            sum -= input[idx + i * cols + j] * kernel[(i + halfKernel) * kernelDim + (j + halfKernel)];
                        }
                        else {
                            sum += input[idx + i * cols + j] * kernel[(i + halfKernel) * kernelDim + (j + halfKernel)];
                        }
                    }
                }
            }
            if (sobelFlag) {
                mask[idx] = sum < 0 ? 1 : 0;
            }
            output[idx] = static_cast<uchar>(sum / reduction);
        }
    }
}

__global__ void threshold(const uchar* input, uchar* output, const int rows, const int cols, const int lowerBound) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;
    const int MAX_UCHAR = 255;
    const int MIN_UCHAR = 0;

    if (row < rows && col < cols) {
        output[idx] = input[idx] >= lowerBound ? MAX_UCHAR : MIN_UCHAR;
    }
}

__global__ void sobelGradientMagnitude(uchar* output, const uchar* outputGx, const uchar* outputGy, const uchar* maskGx, const uchar* maskGy,
                    const int rows, const int cols) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;

    if (row < rows && col < cols) {
        output[idx] = static_cast<uchar>(sqrt(pow(maskGx[idx]
            == 1 ? -static_cast<float>(outputGx[idx]) : static_cast<float>(outputGx[idx]), 2)
            + pow(maskGy[idx] == 1 ? -static_cast<float>(outputGy[idx]) : static_cast<float>(outputGy[idx]), 2)));
    }
}

GPUImageProcessor::GPUImageProcessor() {
    printf("Not implemented yet");
}

void GPUImageProcessor::GPUImageThreshold(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    uchar* d_input, * d_output;
    const int size = rows * cols;

    const uchar lowerBound = 220;

    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_output, size);

    hipMemcpy(d_input, input.ptr(), size, hipMemcpyHostToDevice);

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    threshold << <blocks, threads >> > (d_input, d_output, rows, cols, lowerBound);
    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void GPUImageProcessor::preProcess(const Mat& frame, Mat& output) {  
    Mat gray(frame.rows, frame.cols, CV_8UC1);
    Mat smooth(frame.rows, frame.cols, CV_8UC1);
    Mat thresh(frame.rows, frame.cols, CV_8UC1);

    std::cout << std::endl;
    Start();
    GPUImageGray(frame, gray);
    Stop();
    printTime("GPU Gray");
    
    Start();
    GPUImageSmooth(gray, smooth);
    Stop();
    printTime("GPU Smooth");

    Start();
    GPUImageThreshold(smooth, thresh);
    Stop();
    printTime("GPU Thresh");

    Start();
    GPUImageEdge(thresh, output);
    Stop();
    printTime("GPU Edge");
}

void GPUImageProcessor::GPUImageEdge(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    uchar* d_input, *d_output, *d_kernelGx, *d_kernelGy, * d_maskGx, *d_maskGy, *d_outputGx, *d_outputGy;
    const int size = rows * cols;
    const int reduction = 1;

    uchar Gx[9] = { 1, 0, 1, 2, 0, 2, 1, 0, 1 };
    uchar Gy[9] = { 1, 2, 1, 0, 0, 0, 1, 2, 1 };

    Mat maskGx(rows, cols, CV_8UC1);
    Mat maskGy(rows, cols, CV_8UC1);
    Mat outputGx(rows, cols, CV_8UC1);
    Mat outputGy(rows, cols, CV_8UC1);
    Mat kernelGx(3, 3, CV_8UC1, Gx);
    Mat kernelGy(3, 3, CV_8UC1, Gy);

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    // Gx
    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_kernelGx, 9);
    hipMalloc<unsigned char>(&d_maskGx, size);
    hipMalloc<unsigned char>(&d_outputGx, size);
    hipMemcpy(d_input, input.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernelGx, kernelGx.ptr(), 9, hipMemcpyHostToDevice);

    convolution << <blocks, threads >> > (d_input, d_outputGx, d_kernelGx, 3, rows, cols, reduction, 1, d_maskGx);
    //hipDeviceSynchronize();
    
    hipMemcpy(outputGx.ptr(), d_outputGx, size, hipMemcpyDeviceToHost);
    hipMemcpy(maskGx.ptr(), d_maskGx, size, hipMemcpyDeviceToHost);
    hipFree(d_kernelGx);

    // Gy
    hipMalloc<unsigned char>(&d_kernelGy, 9);
    hipMalloc<unsigned char>(&d_maskGy, size);
    hipMalloc<unsigned char>(&d_outputGy, size);

    hipMemcpy(d_kernelGy, kernelGy.ptr(), 9, hipMemcpyHostToDevice);

    convolution <<<blocks, threads >>> (d_input, d_outputGy, d_kernelGy, 3, rows, cols, reduction, 2, d_maskGy);
    hipDeviceSynchronize();

    hipMemcpy(outputGy.ptr(), d_outputGy, size, hipMemcpyDeviceToHost);
    hipMemcpy(maskGy.ptr(), d_maskGy, size, hipMemcpyDeviceToHost);

    hipFree(d_kernelGy);
    hipFree(d_input);

    // sqrt(Gx^2 + Gy^2
    hipMalloc(&d_output, size);

    sobelGradientMagnitude << <blocks, threads >> > (d_output, d_outputGx, d_outputGy, d_maskGx, d_maskGy, rows, cols);
    hipDeviceSynchronize();

    /*
    // Process Gx and Gy
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            output.at<uchar>(i, j) = static_cast<uchar>(sqrt(pow(maskGx.at<uchar>(i, j) 
                == 1 ? -static_cast<float>(outputGx.at<uchar>(i, j)) : static_cast<float>(outputGx.at<uchar>(i, j)), 2) 
                + pow(maskGy.at<uchar>(i, j) == 1 ? -static_cast<float>(outputGy.at<uchar>(i, j)) : static_cast<float>(outputGy.at<uchar>(i, j)), 2)));
        }
    }
    */

    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_outputGx);
    hipFree(d_outputGy);
    hipFree(d_maskGy);
    hipFree(d_maskGx);
    hipFree(d_input);
}

void GPUImageProcessor::GPUImageGray(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    Mat red(rows, cols, CV_8UC1), green(rows, cols, CV_8UC1), blue(rows, cols, CV_8UC1);
    uchar* d_red, * d_green, * d_blue, * d_gray;

    const int size = red.step * red.rows;

    dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    dim3 threads(32, 32);

    threeChannelToArray(input, red, green, blue);

    Start();
    hipMalloc<unsigned char>(&d_gray, size);

    hipMalloc<unsigned char>(&d_red, size);
    hipMemcpy(d_red, red.ptr(), size, hipMemcpyHostToDevice);
    colorToGray << <blocks, threads >> > (d_red, d_gray, red.cols, red.rows, 1);

    hipMalloc<unsigned char>(&d_green, size);
    hipMemcpy(d_green, green.ptr(), size, hipMemcpyHostToDevice);
    colorToGray << <blocks, threads >> > (d_green, d_gray, red.cols, red.rows, 2);

    hipMalloc<unsigned char>(&d_blue, size);    
    hipMemcpy(d_blue, blue.ptr(), size, hipMemcpyHostToDevice);
    colorToGray << <blocks, threads >> > (d_blue, d_gray, red.cols, red.rows, 3);
    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_gray, size, hipMemcpyDeviceToHost);
    hipFree(d_red);
    hipFree(d_blue);
    hipFree(d_green);
    hipFree(d_gray);
}
void GPUImageProcessor::GPUImageSmooth(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    const int size = rows * cols;
    
    uchar data[9] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
    Mat kernel(3, 3, CV_8UC1, data);
    const float reduction = 16;
    uchar* d_input, * d_output, * d_kernel;

    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_output, size);
    hipMalloc<unsigned char>(&d_kernel, 9);

    hipMemcpy(d_input, input.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.ptr(), 9, hipMemcpyHostToDevice);

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    convolution << <blocks, threads >> > (d_input, d_output, d_kernel, 3, rows, cols, reduction);
    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

void GPUImageProcessor::houghLineTransform(Mat& frame, Mat& result) {
    printf("Not Implemented yet");
}

void GPUImageProcessor::threeChannelToArray(const Mat& frame, Mat& red, Mat& blue, Mat& green) {
    Mat bgrComponents[3];
    split(frame, bgrComponents);

    for (int i = 0; i < frame.rows; i++) {
        for (int j = 0; j < frame.cols; j++) {
            red.at<uchar>(i, j) = bgrComponents[0].at<uchar>(i,j);
            blue.at<uchar>(i, j) = bgrComponents[1].at<uchar>(i, j);
            green.at<uchar>(i, j) = bgrComponents[2].at<uchar>(i, j);
        }
    }
}

