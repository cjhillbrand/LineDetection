#include "hip/hip_runtime.h"
#include "GPUImageProcessor.h"
#include "hip/hip_runtime.h"
#include ""

const int SIZE = 1024; // Number of threads per block (32 x 32)

// Kernel Functions...
__global__ void colorToGray(const uchar* input, uchar* d_gray, const int rowLength, const int colLength, const int colorFlag) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * rowLength + col;
    const int tid = threadIdx.x * blockDim.x + threadIdx.y;
   
    float grayval = 0;
    if (row < colLength && col < rowLength) {
        if (colorFlag == 1) {   
            d_gray[idx] += static_cast<uchar>(0.21 * input[idx]); //  Red
        }
        else if (colorFlag == 2) {
            d_gray[idx] += static_cast<uchar>(0.07 * input[idx]); //  Blue
        }
        else {
            d_gray[idx] += static_cast<uchar>(0.72 * input[idx]); //  Green
        }
    }
}

__constant__ int halfKernel = 1;

__global__ void convolution(const uchar* input, uchar* output, const uchar* kernel, const int kernelDim,
                             const int rows, const int cols, const float reduction, const int sobelFlag = 0, uchar* mask = 0) {

    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x * blockDim.x + threadIdx.y;
    const int idx = row * cols + col;
    float sum = 0;

    if (row < rows && col < cols) {
        if (row < halfKernel || col < halfKernel || row + halfKernel > rows || col + halfKernel > cols) {
            // global edge case, do nothing for now
            output[idx] = input[idx];
        }
        else {
            // i iterates over the columns, j iterates over the rows in inner block
            if (kernelDim == 9 && !sobelFlag) {
                sum += input[idx - 1 * cols - 1] * kernel[(- 1 + halfKernel) * kernelDim + (- 1 + halfKernel)];  // -1 -1
                sum += input[idx * cols - 1] * kernel[(halfKernel) * kernelDim + (-1 + halfKernel)];             //  0 -1
                sum += input[idx + 1 * cols - 1] * kernel[(1 + halfKernel) * kernelDim + (-1 + halfKernel)];     //  1 -1
                sum += input[idx -1 * cols] * kernel[(-1 + halfKernel) * kernelDim + (halfKernel)];              // -1  0
                sum += input[idx * cols] * kernel[(halfKernel) * kernelDim + (halfKernel)];                      //  0  0
                sum += input[idx + 1 * cols] * kernel[(1 + halfKernel) * kernelDim + (halfKernel)];              //  1  0
                sum += input[idx - 1 * cols + 1] * kernel[(-1 + halfKernel) * kernelDim + (1 + halfKernel)];     // -1  1
                sum += input[idx * cols + 1] * kernel[(halfKernel) * kernelDim + (1 + halfKernel)];              //  0  1
                sum += input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];      //  1  1
            }
            else if (sobelFlag) {
                if (sobelFlag == 1) {
                    sum -= input[idx - 1 * cols - 1] * kernel[(-1 + halfKernel) * kernelDim + (-1 + halfKernel)];
                } else {
                    sum += input[idx - 1 * cols - 1] * kernel[(-1 + halfKernel) * kernelDim + (-1 + halfKernel)];
                }
                if (sobelFlag == 2) {
                    sum += input[idx + 0 * cols - 1] * kernel[(halfKernel)*kernelDim + (-1 + halfKernel)];
                }
                sum += input[idx + 1 * cols - 1] * kernel[(1 + halfKernel) * kernelDim + (-1 + halfKernel)];   
                if (sobelFlag == 1) {
                    sum -= input[idx - 1 * cols] * kernel[(-1 + halfKernel) * kernelDim + (halfKernel)];
                }
                if (sobelFlag == 1) {
                    sum += input[idx + 1 * cols] * kernel[(1 + halfKernel) * kernelDim + (halfKernel)];
                }
                sum -= input[idx - 1 * cols + 1] * kernel[(-1 + halfKernel) * kernelDim + (1 + halfKernel)];
                if (sobelFlag == 2) {
                    sum -= input[idx + 0 * cols + 1] * kernel[(halfKernel)*kernelDim + (1 + halfKernel)];
                }
                if (sobelFlag == 2) {
                    sum -= input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];
                } else {
                    sum += input[idx + 1 * cols + 1] * kernel[(1 + halfKernel) * kernelDim + (1 + halfKernel)];
                }
            }
            // TODO potentially remove this section
            else {
                for (int i = -halfKernel; i <= halfKernel; i++) {
                    for (int j = -halfKernel; j <= halfKernel; j++) {
                        // Gx kernel when sobel == 1, Gy kernel when sobel == 2
                        if ((sobelFlag == 1 && j == -halfKernel) || (sobelFlag == 2 && i == halfKernel)) {
                            sum -= input[idx + i * cols + j] * kernel[(i + halfKernel) * kernelDim + (j + halfKernel)];
                        }
                        else {
                            sum += input[idx + i * cols + j] * kernel[(i + halfKernel) * kernelDim + (j + halfKernel)];
                        }
                    }
                }
            }
            if (sobelFlag) {
                mask[idx] = sum < 0 ? 1 : 0;
            }
            output[idx] = static_cast<uchar>(sum / reduction);
        }
    }
}

__global__ void threshold(const uchar* input, uchar* output, const int rows, const int cols, const int lowerBound) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;
    const int MAX_UCHAR = 255;
    const int MIN_UCHAR = 0;

    if (row < rows && col < cols) {
        output[idx] = input[idx] >= lowerBound ? MAX_UCHAR : MIN_UCHAR;
    }
}

__global__ void sobelGradientMagnitude(uchar* output, const uchar* outGx, const uchar* outGy, const uchar* maskGx, const uchar* maskGy,
                    const int rows, const int cols) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;
    const int tid = threadIdx.x * blockDim.x + threadIdx.y;
    
    __shared__ uchar outputGx[SIZE];
    __shared__ uchar outputGy[SIZE];
    
    outputGx[tid] = outGx[idx];
    outputGy[tid] = outGy[idx];

    __syncthreads();
    
    if (row < rows && col < cols) {
        
        output[idx] = static_cast<uchar>(sqrt(pow(maskGx[idx]
            == 1 ? -static_cast<float>(outputGx[tid]) : static_cast<float>(outputGx[tid]), 2)
            + pow(maskGy[idx] == 1 ? -static_cast<float>(outputGy[tid]) : static_cast<float>(outputGy[tid]), 2)));
    }
}

__global__ void TCTOA(const uchar* bgrComponent, uchar* output, const int rows, const int cols) {
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx = row * cols + col;
    const int tid = threadIdx.x * blockDim.x + threadIdx.y;

    __shared__ uchar bgr[SIZE];
    bgr[tid] = bgrComponent[idx];

    __syncthreads();

    output[idx] = bgr[tid];
}

GPUImageProcessor::GPUImageProcessor(std::string filename) : ImageProcessor(filename ) {}

void GPUImageProcessor::preProcess(const Mat& frame, Mat& output) {  
   
    Mat gray(frame.rows, frame.cols, CV_8UC1);
    Mat smooth(frame.rows, frame.cols, CV_8UC1);
    Mat thresh(frame.rows, frame.cols, CV_8UC1);
    
    GPUImageGray(frame, gray);
    GPUImageSmooth(gray, smooth);
    GPUImageThreshold(smooth, thresh);
    GPUImageEdge(thresh, output);
}

void GPUImageProcessor::GPUImageThreshold(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    uchar* d_input, * d_output;
    const int size = rows * cols;
    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    //220
    const uchar lowerBound = 230;

    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_output, size);

    hipMemcpyAsync(d_input, input.ptr(), size, hipMemcpyHostToDevice);

    threshold << <blocks, threads >> > (d_input, d_output, rows, cols, lowerBound);
    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void GPUImageProcessor::GPUImageEdge(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    uchar* d_input, *d_output, *d_kernelGx, *d_kernelGy, * d_maskGx, *d_maskGy, *d_outputGx, *d_outputGy;
    const int size = rows * cols;
    const int reduction = 1;

    uchar Gx[9] = { 1, 0, 1, 2, 0, 2, 1, 0, 1 };
    uchar Gy[9] = { 1, 2, 1, 0, 0, 0, 1, 2, 1 };

    Mat maskGx(rows, cols, CV_8UC1);
    Mat maskGy(rows, cols, CV_8UC1);
    Mat outputGx(rows, cols, CV_8UC1);
    Mat outputGy(rows, cols, CV_8UC1);
    Mat kernelGx(3, 3, CV_8UC1, Gx);
    Mat kernelGy(3, 3, CV_8UC1, Gy);

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    const int NUM_OF_STREAMS = 2;
    hipStream_t streams[NUM_OF_STREAMS];
   
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    // Gx
    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_kernelGx, 9);
    hipMalloc<unsigned char>(&d_maskGx, size);
    hipMalloc<unsigned char>(&d_outputGx, size);
    hipMemcpyAsync(d_input, input.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_kernelGx, kernelGx.ptr(), 9, hipMemcpyHostToDevice);
    
    convolution << <blocks, threads, 0, streams[0] >> > (d_input, d_outputGx, d_kernelGx, 3, rows, cols, reduction, 1, d_maskGx);

    hipMemcpyAsync(outputGx.ptr(), d_outputGx, size, hipMemcpyDeviceToHost);
    hipMemcpyAsync(maskGx.ptr(), d_maskGx, size, hipMemcpyDeviceToHost);

    hipFree(d_kernelGx);

    // Gy
    hipMalloc<unsigned char>(&d_kernelGy, 9);
    hipMalloc<unsigned char>(&d_maskGy, size);
    hipMalloc<unsigned char>(&d_outputGy, size);

    hipMemcpyAsync(d_kernelGy, kernelGy.ptr(), 9, hipMemcpyHostToDevice);
 

    convolution <<<blocks, threads, 0, streams[1] >>> (d_input, d_outputGy, d_kernelGy, 3, rows, cols, reduction, 2, d_maskGy);
    hipDeviceSynchronize();

    hipMemcpyAsync(outputGy.ptr(), d_outputGy, size, hipMemcpyDeviceToHost);
    hipMemcpyAsync(maskGy.ptr(), d_maskGy, size, hipMemcpyDeviceToHost);

    hipFree(d_kernelGy);
    hipFree(d_input);

    hipMalloc(&d_output, size);

    // sqrt(Gx^2 + Gy^2
    sobelGradientMagnitude << <blocks, threads >> > (d_output, d_outputGx, d_outputGy, d_maskGx, d_maskGy, rows, cols);
    hipDeviceSynchronize();


    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_outputGx);
    hipFree(d_outputGy);
    hipFree(d_maskGy);
    hipFree(d_maskGx);
    hipFree(d_input);
}

void GPUImageProcessor::GPUImageGray(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    Mat red(rows, cols, CV_8UC1), green(rows, cols, CV_8UC1), blue(rows, cols, CV_8UC1);
    uchar* d_red, * d_green, * d_blue, * d_gray;

    const int size = red.step * red.rows;
    const int NUM_OF_STREAMS = 3;
    hipStream_t streams[NUM_OF_STREAMS];

    dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    dim3 threads(32, 32);

    threeChannelToArray(input, red, green, blue);

    hipMalloc<unsigned char>(&d_gray, size);
    hipMalloc<unsigned char>(&d_red, size);
    hipMalloc<unsigned char>(&d_green, size);
    hipMalloc<unsigned char>(&d_blue, size);

    hipMemcpyAsync(d_red, red.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_green, green.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_blue, blue.ptr(), size, hipMemcpyHostToDevice);

    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    hipStreamCreate(&streams[2]);

    colorToGray << <blocks, threads, 0, streams[0] >> > (d_red, d_gray, red.cols, red.rows, 1);
    colorToGray << <blocks, threads, 0, streams[1] >> > (d_green, d_gray, red.cols, red.rows, 2);
    colorToGray << <blocks, threads, 0, streams[2] >> > (d_blue, d_gray, red.cols, red.rows, 3);
    hipDeviceSynchronize();
    
    hipMemcpy(output.ptr(), d_gray, size, hipMemcpyDeviceToHost);
    hipFree(d_red);
    hipFree(d_blue);
    hipFree(d_green);
    hipFree(d_gray);
}
void GPUImageProcessor::GPUImageSmooth(const Mat& input, Mat& output) {
    const int rows = input.rows, cols = input.cols;
    const int size = rows * cols;
    
    uchar data[9] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
    Mat kernel(3, 3, CV_8UC1, data);
    const float reduction = 16;
    uchar* d_input, * d_output, * d_kernel;

    hipMalloc<unsigned char>(&d_input, size);
    hipMalloc<unsigned char>(&d_output, size);
    hipMalloc<unsigned char>(&d_kernel, 9);

    hipMemcpyAsync(d_input, input.ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_kernel, kernel.ptr(), 9, hipMemcpyHostToDevice);

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    convolution << <blocks, threads >> > (d_input, d_output, d_kernel, 3, rows, cols, reduction);
    hipDeviceSynchronize();

    hipMemcpy(output.ptr(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

void GPUImageProcessor::houghLineTransform(Mat& frame, Mat& result) {
    printf("Not Implemented yet");
}

void GPUImageProcessor::threeChannelToArray(const Mat& frame, Mat& red, Mat& blue, Mat& green) {
    Mat bgrComponents[3];

    const int size = frame.rows * frame.cols;
    const int rows = frame.rows, cols = frame.cols;

    const dim3 blocks(ceil((double)cols / 32.0), ceil((double)rows / 32.0));
    const dim3 threads(32, 32);

    const int NUM_OF_STREAMS = 3;
    hipStream_t streams[NUM_OF_STREAMS];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    hipStreamCreate(&streams[2]);

    split(frame, bgrComponents);
 
    uchar *d_redComponent, *d_blueComponent, *d_greenComponent, *d_red, *d_blue, *d_green;

    hipMalloc<unsigned char>(&d_red, size);
    hipMalloc<unsigned char>(&d_blue, size);
    hipMalloc<unsigned char>(&d_green, size);
    hipMalloc<unsigned char>(&d_redComponent, size);
    hipMalloc<unsigned char>(&d_blueComponent, size);
    hipMalloc<unsigned char>(&d_greenComponent, size);

   
    hipMemcpyAsync(d_redComponent, bgrComponents[0].ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_blueComponent, bgrComponents[1].ptr(), size, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_greenComponent, bgrComponents[2].ptr(), size, hipMemcpyHostToDevice);

    TCTOA << <blocks, threads, 0, streams[0] >> > (d_redComponent, d_red, rows, cols);
    TCTOA << <blocks, threads, 0, streams[1] >> > (d_blueComponent, d_blue, rows, cols);
    TCTOA << <blocks, threads, 0, streams[2] >> > (d_greenComponent, d_green, rows, cols);
    hipDeviceSynchronize();

    hipMemcpyAsync(red.ptr(), d_red, size, hipMemcpyDeviceToHost);
    hipMemcpyAsync(blue.ptr(), d_blue, size, hipMemcpyDeviceToHost);
    hipMemcpyAsync(green.ptr(), d_green, size, hipMemcpyDeviceToHost);

    hipFree(d_red);
    hipFree(d_blue);
    hipFree(d_green);
    hipFree(d_redComponent);
    hipFree(d_blueComponent);
    hipFree(d_greenComponent);
    
}

